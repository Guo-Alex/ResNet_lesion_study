#include "hip/hip_runtime.h"
#include <iostream>
#include "caffe/layers/quantize.hpp"

namespace caffe {

template<typename Dtype>
__global__ void Quantize3(const int n, const int H, const int W,
		const Dtype* in, Dtype* out) {
	CUDA_KERNEL_LOOP(index, n)
	{
		int w = index % W;
		int h = index / W % H;
		int n = index / W / H;
		float temp, c;

		temp = in[index] / 1;
		c = (int) temp;
		if (temp >= 0) {
			temp = temp - c >= 0.5 ? c + 1 : c;
		} else {
			temp = temp - c <= -0.5 ? 1 - c : -c;
		}
		if (temp > 4)
			temp = 4;
		out[((n * 3 + 0) * H + h) * W + w] = temp;

		temp = in[index] / 2;
		c = (int) temp;
		if (temp >= 0) {
			temp = temp - c >= 0.5 ? c + 1 : c;
		} else {
			temp = temp - c <= -0.5 ? 1 - c : -c;
		}
		if (temp > 4)
			temp = 4;
		out[((n * 3 + 1) * H + h) * W + w] = temp;

		temp = in[index] / 4;
		c = (int) temp;
		if (temp >= 0) {
			temp = temp - c >= 0.5 ? c + 1 : c;
		} else {
			temp = temp - c <= -0.5 ? 1 - c : -c;
		}
		if (temp > 4)
			temp = 4;
		out[((n * 3 + 2) * H + h) * W + w] = temp;
	}
}

template<typename Dtype>
void QuantizeLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top) {

	const Dtype* bottom_data = bottom[0]->gpu_data();
	Dtype* top_data = top[0]->mutable_gpu_data();
	const int count = bottom[0]->count();

	Quantize3<Dtype> <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
			count, height_, width_, bottom_data, top_data);
	CUDA_POST_KERNEL_CHECK;

	/*std::ofstream myfile5;
	myfile5.open("example5.txt");
	std::ofstream myfile6;
	myfile6.open("example6.txt");
	for (int i=0; i<count/2; i++) {
		myfile5 << (top[0]->cpu_data())[i] << ' ';
		myfile6 << (top[0]->cpu_data())[i+count/2] << ' ';
	}
	myfile5.close();
	myfile6.close();*/

}

INSTANTIATE_LAYER_GPU_FUNCS(QuantizeLayer);
}  // namespace caffe
